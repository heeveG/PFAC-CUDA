#include <iostream>
#include "headers/util.cuh"
#include "hip/hip_runtime_api.h"

int main() {
    // setup the device
    check(hipSetDevice(0));
    hipDeviceProp_t deviceProp{};
    check(hipGetDeviceProperties(&deviceProp, 0));
    std::cout << "GPU used : " << deviceProp.name << std::endl;

    // read input data
    std::basic_string<char, std::char_traits<char>, managed_allocator<char>> input;
//    std::string input;
    std::vector<std::string> files{
            "../data/2600-0.txt", "../data/2701-0.txt", "../data/35-0.txt", "../data/84-0.txt", "../data/8800.txt",
            "../data/pg1727.txt", "../data/pg55.txt", "../data/pg6130.txt", "../data/pg996.txt", "../data/1342-0.txt"
    };
    read_data(input, files);

    // build trie
    std::unordered_map<std::string, int> patternIdMap;
    std::vector<trie, managed_allocator<trie>> nodes(1 << 17);

    do_trie(nodes, input, patternIdMap);

    int numPatterns = patternIdMap.size();
    auto root = nodes.data();

    const int blockSize = 256;
    const int numBlocks = 1024;
//    const int numBlocks = ((int) input.size() + blockSize - 1) / blockSize;

//    hipProfilerStart();
    int *matches = (int *) malloc(numPatterns * sizeof(int));
    int *d_matched;
//    char *d_input;
//
//    check(hipMalloc(&d_input, input.size()));
    check(hipMalloc(&d_matched, numPatterns * sizeof(int)));
//    check(hipMemcpy(d_input, input.data(), input.size(), hipMemcpyHostToDevice));

    check(hipMemPrefetchAsync(nodes.data(), 80564 * sizeof(trie), 0));
    check(hipMemPrefetchAsync(input.data(), input.size(), 0));

    // perform matching
    float matchingTime = cudaEventProfile([&]() {
        matchWords<<<numBlocks, blockSize>>>(input.data(), d_matched, root, input.size());
    });

    check(hipMemcpy(matches, d_matched, numPatterns * sizeof(int), hipMemcpyDeviceToHost));
//    hipProfilerStop();
    // validate results
    if (validateResult("../validation/results.csv", patternIdMap, matches))
        std::cout << "Matching completed successfully in: " << matchingTime << " ms. " << std::endl;
    else
        std::cout << "Invalid results" << std::endl;

//    do_trie(input, deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor >> 10, 1 << 10);

    free(matches);
    hipFree(d_matched);
//    hipFree(d_input);
    return 0;
}

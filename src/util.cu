#include "hip/hip_runtime.h"
//
// Created by heeve on 09.05.22.
//

#include <vector>
#include "../headers/util.cuh"

__host__ __device__
int index_of(char c) {
    if (c >= 'a' && c <= 'z') return c - 'a';
    if (c >= 'A' && c <= 'Z') return c - 'A';
    return -1;
};

__global__ void matchWords(const char *str, int *matched, trie *root, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const trie *node;

    for (int iter = i; iter < size; iter += stride) {
        node = root;
        while (i < size) {
            int index = index_of(str[i]);

            if (index == -1)
                break;

            node = node->next[index].ptr;
            if (node == nullptr)
                break;

            if (node->id != -1)
                atomicAdd(&matched[node->id], 1);

            ++i;
        }

        i = iter + stride;
    }
}

__host__
int host_make_trie(trie &root, trie *&bump, const char *begin, const char *end,
                   std::unordered_map<std::string, int> &patternIdMap) {
    int patternId = 0;
    int counter = 0;
    std::string word;
    auto n = &root;
    for (auto pc = begin; pc != end; ++pc) {
        auto const index = index_of(*pc);
        if (index == -1) {
            if (n != &root) {
                if (n->id == -1) { // $todo account for data race in multithreaded
                    n->id = patternId++;
                    patternIdMap.insert(std::make_pair(word, n->id));
                } else
                    counter++;
                n = &root;
            }
            word = "";
            continue;
        }
        word += *pc;
        if (n->next[index].ptr == nullptr)
            n->next[index].ptr = bump++;
        n = n->next[index].ptr;
    }

    std::cout << "same patterns: " << counter << std::endl;
    return patternId;
}

// $TODO - add Turing support
//__host__ __device__
//void device_make_trie(trie &root,
//                   simt::std::atomic<trie *> &bump,
//                   const char *begin, const char *end,
//                   unsigned index,
//                   unsigned domain) {
//
//    auto const size = end - begin;
//    auto const stride = (size / domain + 1);
//
//    auto off = minimum(size, stride * index);
//    auto const last = minimum(size, off + stride);
//
//    for (char c = begin[off]; off < size && off != last && c != 0 && index_of(c) != -1; ++off, c = begin[off]);
//    for (char c = begin[off]; off < size && off != last && c != 0 && index_of(c) == -1; ++off, c = begin[off]);
//
//    trie *n = &root;
//    for (char c = begin[off];; ++off, c = begin[off]) {
//        auto const index = off >= size ? -1 : index_of(c);
//        if (index == -1) {
//            if (n != &root) {
//                n->count.fetch_add(1, simt::std::memory_order_relaxed);
//                n = &root;
//            }
//            //end of last word?
//            if (off >= size || off > last)
//                break;
//            else
//                continue;
//        }
//        if (n->next[index].ptr.load(simt::memory_order_acquire) == nullptr) {
//            if (n->next[index].flag.test_and_set(simt::std::memory_order_relaxed))
//                n->next[index].ptr.wait(nullptr, simt::std::memory_order_acquire);
//            else {
//                auto next = bump.fetch_add(1, simt::std::memory_order_relaxed);
//                n->next[index].ptr.store(next, simt::std::memory_order_release);
//                n->next[index].ptr.notify_all();
//            }
//        }
//        n = n->next[index].ptr.load(simt::std::memory_order_relaxed);
//    }
//}
//
//__global__
//void gpu_make_trie(trie *t, simt::std::atomic<trie *> *bump, const char *begin, const char *end) {
//
//    auto const index = blockDim.x * blockIdx.x + threadIdx.x;
//    auto const domain = gridDim.x * blockDim.x;
//    make_trie(*t, *bump, begin, end, index, domain);
//
//}

bool validateResult(const char *csvPath, std::unordered_map<std::string, int> &patternIdMap, int *matches) {
    std::unordered_map<std::string, int> validMatches;

    // read valid results
    std::ifstream fin(csvPath);
    if (!fin.good()) {
        std::cerr << "Error opening '" << "'. Bailing out." << std::endl;
        exit(-1);
    }

    std::string pattern;
    int count;

    while (fin >> pattern >> count)
        validMatches[pattern] = count;

    fin.close();

    // validate
    for (const auto & match : validMatches)
        if (matches[patternIdMap.at(match.first)] != match.second)
            return false;
    return true;
}
